#include "hip/hip_runtime.h"
#include <stdint.h>
#include <stdio.h>
#include "drillx.h"
#include "equix/include/equix.h"
#include "hashx/include/hashx.h"
#include "equix/src/context.h"
#include "equix/src/solver.h"
#include "equix/src/solver_heap.h"
#include "hashx/src/context.h"

const int BATCH_SIZE = 2048;

extern "C" void hash(uint8_t *challenge, uint8_t *nonce, uint64_t *out) {
    // Allocate pinned memory for ctxs and hash_space
    hashx_ctx** ctxs;
    uint64_t** hash_space;

    hipHostMalloc(&ctxs, BATCH_SIZE * sizeof(hashx_ctx*));
    hipHostMalloc(&hash_space, BATCH_SIZE * sizeof(uint64_t*));

    for (int i = 0; i < BATCH_SIZE; i++) {
        hipMalloc(&hash_space[i], INDEX_SPACE * sizeof(uint64_t));
    }

    // Prepare seed and hash contexts
    uint8_t seed[40];
    memcpy(seed, challenge, 32);
    for (int i = 0; i < BATCH_SIZE; i++) {
        uint64_t nonce_offset = *((uint64_t*)nonce) + i;
        memcpy(seed + 32, &nonce_offset, 8);
        ctxs[i] = hashx_alloc(HASHX_INTERPRETED);
        if (!ctxs[i] || !hashx_make(ctxs[i], seed, 40)) {
            hipHostFree(ctxs);
            return;
        }
    }

    // Launch kernel to parallelize hashx operations
    dim3 threadsPerBlock(1024); // 256 threads per block
    dim3 blocksPerGrid((65536 * BATCH_SIZE + threadsPerBlock.x - 1) / threadsPerBlock.x); // enough blocks to cover batch
    do_hash_stage0i<<<blocksPerGrid, threadsPerBlock>>>(ctxs, hash_space);
    

    // Copy hashes back to cpu
    for (int i = 0; i < BATCH_SIZE; i++) {
        hipMemcpy(out + i * INDEX_SPACE, hash_space[i], INDEX_SPACE * sizeof(uint64_t), hipMemcpyDeviceToHost);
    }

    // Free memory
    for (int i = 0; i < BATCH_SIZE; i++) {
        hashx_free(ctxs[i]);
        hipFree(hash_space[i]);
    }
    hipHostFree(hash_space);
    hipHostFree(ctxs);
}

__global__ void do_hash_stage0i(hashx_ctx** ctxs, uint64_t** hash_space) {
    uint32_t item = blockIdx.x * blockDim.x + threadIdx.x;
    uint32_t batch_idx = item / INDEX_SPACE;
    uint32_t i = item % INDEX_SPACE;
    if (batch_idx < BATCH_SIZE) {
        hash_stage0i(ctxs[batch_idx], hash_space[batch_idx], i);
    }
}

extern "C" void solve_all_stages(uint64_t *hashes, uint8_t *out, uint32_t *sols, int num_sets) {
    // Allocate device memory
    uint64_t *d_hashes;
    solver_heap *d_heaps;
    equix_solution *d_solutions;
    uint32_t *d_num_sols;

    hipMalloc(&d_hashes, num_sets * INDEX_SPACE * sizeof(uint64_t));
    hipMalloc(&d_heaps, num_sets * sizeof(solver_heap));
    hipMalloc(&d_solutions, num_sets * EQUIX_MAX_SOLS * sizeof(equix_solution));
    hipMalloc(&d_num_sols, num_sets * sizeof(uint32_t));

    // Allocate pinned host memory
    equix_solution *h_solutions;
    uint32_t *h_num_sols;
    hipHostAlloc(&h_solutions, num_sets * EQUIX_MAX_SOLS * sizeof(equix_solution), hipHostMallocDefault);
    hipHostAlloc(&h_num_sols, num_sets * sizeof(uint32_t), hipHostMallocDefault);

    // Copy input data to device
    hipMemcpy(d_hashes, hashes, num_sets * INDEX_SPACE * sizeof(uint64_t), hipMemcpyHostToDevice);

    // Launch kernel
    int threadsPerBlock = 256;
    int blocksPerGrid = (num_sets + threadsPerBlock - 1) / threadsPerBlock;
    solve_all_stages_kernel<<<blocksPerGrid, threadsPerBlock>>>(d_hashes, d_heaps, d_solutions, d_num_sols);

    // Copy results back to host using pinned memory
    hipMemcpy(h_solutions, d_solutions, num_sets * EQUIX_MAX_SOLS * sizeof(equix_solution), hipMemcpyDeviceToHost);
    hipMemcpy(h_num_sols, d_num_sols, num_sets * sizeof(uint32_t), hipMemcpyDeviceToHost);

    // Process results
    for (int i = 0; i < num_sets; i++) {
        sols[i] = h_num_sols[i];
        if (h_num_sols[i] > 0) {
            memcpy(out + i * sizeof(equix_solution), &h_solutions[i * EQUIX_MAX_SOLS], sizeof(equix_solution));
        }
    }

    // Free device memory
    hipFree(d_hashes);
    hipFree(d_heaps);
    hipFree(d_solutions);
    hipFree(d_num_sols);

    // Free pinned host memory
    hipHostFree(h_solutions);
    hipHostFree(h_num_sols);
}